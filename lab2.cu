#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define MAX_RAND 10

// Device Code, 1 dimensional
__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


// Host Code
// Sequential CPU implementation of vector addition
void vector_add(int* a, int* b, int* result, int n) {
    for (int i = 0; i < n; i++) {
        result[i] = a[i] + b[i];
    }
}

// Generate random numbers from -10 to 10
void random_ints(int* a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = (rand() % (2 * MAX_RAND + 1)) - MAX_RAND; // Creates Random from -10 to 10
}
//-----------------
void printArray(int* vals, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", vals[i]);
    }
    printf("\n");
}
//-----------------

int main(void) {

	// Open a new file for writing
    FILE* deviceToHostTime = fopen("deviceToHostTime.csv", "w");
	FILE* hostToDeviceTime = fopen("hostToDeviceTime.csv", "w");
	FILE* operationTime = fopen("operationTime.csv", "w");

	// Declare variables to hold the start and end times
	clock_t start, end;
	double deviceToHost_time_used, hostToDevice_time_used, mathOperation_time_used;


	for(int N = 32; N<=1073741824; N*=2){
		// Initialize vectors
		int *a, *b, *c;
		int *d_a, *d_b, *d_c;
		int size = N * sizeof(int);

		// Allocate Device memory
		hipMalloc((void **)&d_a,size);
		hipMalloc((void **)&d_b,size);
		hipMalloc((void **)&d_c,size);

		// Allocate Host memory and fill vectors with random ints
		a=(int *)malloc(size); 
		b=(int *)malloc(size); 
		c=(int *)malloc(size);
		random_ints(a,N);
		random_ints(b,N);

		//printf("A = "); printArray(a,N);
		//printf("B = "); printArray(b,N);

		// Execute in Device
		for(int blocksPerGrid = 1; blocksPerGrid<=1024; blocksPerGrid*=2){
			for(int threadsPerBlock = 1; threadsPerBlock<=1024; threadsPerBlock*=2){
				// Host to Device
				start = clock();
				hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
				hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
				end = clock();
				hostToDevice_time_used = ((double) (end - start))/CLOCKS_PER_SEC;
				fprintf(hostToDeviceTime, "%d, %f\n", N, hostToDevice_time_used);

				// Doing the addition
				start = clock();
				add<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_c);
				hipDeviceSynchronize();
				end = clock();
				mathOperation_time_used = ((double) (end - start))/CLOCKS_PER_SEC;
				fprintf(operationTime, "%d, %d,%d,%f\n", N, blocksPerGrid, threadsPerBlock, mathOperation_time_used);

				// Get result back from Drvice to Host
				start = clock();
				hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
				end = clock();
				deviceToHost_time_used = ((double) (end - start))/CLOCKS_PER_SEC;
				fprintf(deviceToHostTime, "%d,%f\n", N, deviceToHost_time_used);
			}
		}
		//printf("C = "); printArray(c,N);

		free(a);
		free(b);
		free(c);
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
	}

	fclose(deviceToHostTime);
	fclose(hostToDeviceTime);
	fclose(operationTime);
	return 0;
}